#include "hip/hip_runtime.h"
/*!
 * Copyright (c) 2015 by Contributors
 * \file upsampling_nearest.cc
 * \brief
 * \author Bing Xu
*/

#include "./deconvolution-inl.h"
#include "./upsampling-inl.h"

namespace mxnet {
namespace op {
template<>
Operator *CreateOp<gpu>(UpSamplingParam param, int dtype) {
  Operator *op = NULL;
  MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
    if (param.sample_type == up_enum::kNearest) {
      op = new UpSamplingNearestOp<gpu, DType>(param);
    } else if (param.sample_type == up_enum::kBilinear) {
      DeconvolutionParam p = DeconvolutionParam();
      int kernel = 2 * param.scale - param.scale % 2;
      int stride = param.scale;
      int pad = static_cast<int>(ceil((param.scale - 1) / 2.));
      p.workspace = param.workspace;
      p.num_group = param.num_filter;
      p.num_filter = param.num_filter;
      p.no_bias =  true;
      int shape[] = {1, 1};
      shape[0] = shape[1] = kernel;
      p.kernel = TShape(shape, shape + 2);
      shape[0] = shape[1] = stride;
      p.stride = TShape(shape, shape + 2);
      shape[0] = shape[1] = pad;
      p.pad = TShape(shape, shape + 2);
      op = new DeconvolutionOp<gpu, DType>(p);
    } else {
      LOG(FATAL) << "Unknown sample type";
    }
  });
  return op;
}

}  // namespace op
}  // namespace mxnet
