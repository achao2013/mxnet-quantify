#include "hip/hip_runtime.h"
/*!
 * Copyright (c) 2015 by Contributors
 * \file activation.cu
 * \brief
 * \author Bing Xu
*/
#include "./activation-inl.h"
#include "./mshadow_op.h"
#if MXNET_USE_CUDNN == 1
#include "./cudnn_activation-inl.h"
#endif

namespace mxnet {
namespace op {
template<>
Operator *CreateOp<gpu>(ActivationParam param, int dtype) {
  Operator *op = NULL;
  // SoftReLU not supported by CUDNN yet
  if (param.act_type == activation::kSoftReLU) {
    MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
      op = new ActivationOp<gpu, mshadow_op::softrelu, mshadow_op::softrelu_grad, DType>();
    })
    op->set_q_method(param.q_method);//add by jzc
	op->set_act_type(param.act_type);
    return op;
  }
  else if(param.act_type == activation::kSign){
	MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
	  op = new ActivationOp<gpu, mshadow_op::binary, mshadow_op::binary_grad, DType>();
	})
	op->set_q_method(param.q_method);//add by jzc
	op->set_act_type(param.act_type);
	return op;
  }
#if MXNET_USE_CUDNN == 1
  MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
    op = new CuDNNActivationOp<DType>(param);
  })
#else
  MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
    switch (param.act_type) {
      case activation::kReLU:
        op = new ActivationOp<gpu, mshadow_op::relu, mshadow_op::relu_grad, DType>();
        break;
      case activation::kSigmoid:
        op = new ActivationOp<gpu, mshadow_op::sigmoid, mshadow_op::sigmoid_grad, DType>();
        break;
      case activation::kTanh:
        op = new ActivationOp<gpu, mshadow_op::tanh, mshadow_op::tanh_grad, DType>();
        break;
      default:
        LOG(FATAL) << "unknown activation";
    }
  })
  op->set_act_type(param.act_type);//add by jzc
#endif  // MXNET_USE_CUDNN
  op->set_q_method(param.q_method);//add by jzc

  return op;
}
}  // namespace op
}  // namespace mxnet

