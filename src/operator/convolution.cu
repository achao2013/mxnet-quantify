/*!
 * Copyright (c) 2015 by Contributors
 * \file convolution.cu
 * \brief
 * \author Bing Xu
*/

#include "./convolution-inl.h"
#include <vector>
#if MXNET_USE_CUDNN == 1
#include "./cudnn_convolution-inl.h"
#endif  // MXNET_USE_CUDNN

namespace mxnet {
namespace op {
template<>
Operator* CreateOp<gpu>(ConvolutionParam param, int dtype,
                        std::vector<TShape> *in_shape,
                        std::vector<TShape> *out_shape,
                        Context ctx) {
  Operator *op = NULL;
#if MXNET_USE_CUDNN == 1
  if (param.dilate[0] == 1 && param.dilate[1] == 1) {
    MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
      op = new CuDNNConvolutionOp<DType>(param, in_shape, out_shape, ctx);
    })
  } else {
    MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
      op = new ConvolutionOp<gpu, DType>(param);
    })
  }
#else
  MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
    op = new ConvolutionOp<gpu, DType>(param);
  })
#endif  // MXNET_USE_CUDNN
  return op;
}



template<>
void mx_xpu_asum<mshadow::gpu,float>(mshadow::Stream<mshadow::gpu>* s,const int n, const float* x, float* y)
{
    hipblasSasum(mshadow::Stream<mshadow::gpu>::GetBlasHandle(s),n,x,1,y);
}
template<>
void mx_xpu_asum<mshadow::gpu,double>(mshadow::Stream<mshadow::gpu>* s,const int n, const double* x, double* y)
{
	hipblasDasum(mshadow::Stream<mshadow::gpu>::GetBlasHandle(s),n,x,1,y);
}
template<>
void mx_xpu_asum<mshadow::cpu,float>(mshadow::Stream<mshadow::cpu>* s,const int n, const float* x, float* y)
{
    *y=cblas_sasum(n,x,1);
}
template<>
void mx_xpu_asum<mshadow::cpu,double>(mshadow::Stream<mshadow::cpu>* s,const int n, const double* x, double* y)
{
	*y=cblas_dasum(n,x,1);
}
template<>
void mx_xpu_asum<mshadow::cpu,mshadow::half::half_t>(mshadow::Stream<mshadow::cpu>* s,const int n, const mshadow::half::half_t* x, mshadow::half::half_t* y){}
template<>
void mx_xpu_asum<mshadow::gpu,mshadow::half::half_t>(mshadow::Stream<mshadow::gpu>* s,const int n, const mshadow::half::half_t* x, mshadow::half::half_t* y){}

template <>
void mx_xpu_scal<gpu,double>(mshadow::Stream<mshadow::gpu>* s,const int N, const double alpha, double *X) {
  hipblasDscal(mshadow::Stream<mshadow::gpu>::GetBlasHandle(s), N, &alpha, X, 1);
}
template <>
void mx_xpu_scal<gpu,float>(mshadow::Stream<mshadow::gpu>* s,const int N, const float alpha, float *X) {
  hipblasSscal(mshadow::Stream<mshadow::gpu>::GetBlasHandle(s), N, &alpha, X, 1);
}
template <>
void mx_xpu_scal<cpu,double>(mshadow::Stream<mshadow::cpu>* s,const int N, const double alpha, double *X) {
  cblas_dscal(N, alpha, X, 1);
}
template <>
void mx_xpu_scal<cpu,float>(mshadow::Stream<mshadow::cpu>* s,const int N, const float alpha, float *X) {
  cblas_sscal(N, alpha, X, 1);
}
template<>
void mx_xpu_scal<mshadow::cpu,mshadow::half::half_t>(mshadow::Stream<mshadow::cpu>* s,const int N, const mshadow::half::half_t alpha, mshadow::half::half_t* X){}
template<>
void mx_xpu_scal<mshadow::gpu,mshadow::half::half_t>(mshadow::Stream<mshadow::gpu>* s,const int N, const mshadow::half::half_t alpha,  mshadow::half::half_t* X){}

}  // namespace op
}  // namespace mxnet

