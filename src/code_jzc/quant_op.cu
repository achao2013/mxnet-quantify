#include "hip/hip_runtime.h"
#include "quant_op.h"


namespace mxnet {
namespace op {
		template<>
		__global__ void print_kernal(float* data)
		{
			printf(" %f %f %f %f %f %f %f\n",data[0],data[1],data[2],data[3],data[4],data[5],data[6]);
		}

		template<>
		__global__ void print_kernal(mshadow::half::half_t* data)
		{
			printf(" %f %f %f %f %f %f %f\n",data[0],data[1],data[2],data[3],data[4],data[5],data[6]);
		}

		template<>
		__global__ void print_kernal(double* data)
		{
			printf(" %lf %lf %lf %lf %lf %lf %lf\n",data[0],data[1],data[2],data[3],data[4],data[5],data[6]);
		}

		template<>
		void print_check<gpu>(float * data,std::string s)
		{
			const char* str=s.c_str();
			printf("%s",str);
			print_kernal<<< 1,1 >>>(data);
		}

		template<>
		void print_check<gpu>(double * data,std::string s)
		{
			const char* str=s.c_str();
			printf("%s",str);
			print_kernal<<< 1,1 >>>(data);
		}

		template<>
		void print_check<gpu>(mshadow::half::half_t * data,std::string s)
		{
			const char* str=s.c_str();
			printf("%s",str);
			print_kernal<<< 1,1 >>>(data);
		}

		template<>
		void get_iter<gpu>(float* data, float& cpu_i)
		{
			hipMemcpy(&cpu_i,data,sizeof(float),hipMemcpyDeviceToHost);

		}

		template<>
		void get_iter<gpu>(double* data, double& cpu_i)
		{
			hipMemcpy(&cpu_i,data,sizeof(double),hipMemcpyDeviceToHost);

		}

		template<>
		void get_iter<gpu>(mshadow::half::half_t* data, mshadow::half::half_t& cpu_i)
		{
			hipMemcpy(&cpu_i,data,sizeof(mshadow::half::half_t),hipMemcpyDeviceToHost);

		}
	}
}
